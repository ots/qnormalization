
#include <hip/hip_runtime.h>
#include <sys/time.h>

class SimpleTimer
{
	struct timeval starttime;
public:
	void start();
	double end(); 
};

void SimpleTimer::start()
{
	gettimeofday(&starttime,0);
}

double SimpleTimer::end()
{
	struct timeval endtime;
	gettimeofday(&endtime,0);

	return (endtime.tv_sec - starttime.tv_sec)*1000.0 + (endtime.tv_usec - starttime.tv_usec)/1000.0;
}
