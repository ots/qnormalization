#include "hip/hip_runtime.h"
#include "stdio.h"
#include "gpuqsort.h"
#include "Qfunc.c"

#include "simpletimer.cu"

#include <algorithm>

#define INDICES_TMP "index_tmp.bin"

// Keep tracks of the data blocks in phase one
template <typename element>
struct BlockSize
{
	unsigned int beg;
	unsigned int end;
	unsigned int orgbeg;
	unsigned int orgend;
	element		 rmaxpiv;
	element		 lmaxpiv;
	element		 rminpiv;
	element		 lminpiv;

	bool		 altered;
	bool		 flip;
	element		 pivot;
};

// Holds parameters to the kernel in phase one
template <typename element>
struct Params
{
	unsigned int from;
	unsigned int end;
	element pivot;
	unsigned int ptr;
	bool last;
};

// Used to perform a cumulative sum between blocks.
// Unnecessary for cards with atomic operations.
// Will be removed when these becomes more common
template <typename element>
struct Length
{
	element maxpiv[MAXBLOCKS];
	element minpiv[MAXBLOCKS];

	unsigned int left[MAXBLOCKS];
	unsigned int right[MAXBLOCKS];
};

// Since we have divided up the kernel in to three
// we need to remember the result of the cumulative sum
// Unnecessary for cards with atomic operations.
// Will be removed when these becomes more common
struct Hist
{
	unsigned int left[(MAXTHREADS)*MAXBLOCKS];
	unsigned int right[(MAXTHREADS)*MAXBLOCKS];
};

struct LQSortParams
{
	unsigned int beg;
	unsigned int end;
	bool flip;
	unsigned int sbsize;
};

#include "gpuqsort_kernels.cu"

#undef THREADS
#define THREADS threads

/**
* The main sort function
* @param data		Data to be sorted
* @param size		The length of the data
* @param timerValue Contains the time it took to sort the data [Optional]
* @returns 0 if successful. For non-zero values, use getErrorStr() for more information about why it failed.
*/
template <typename element>
int GPUQSort<element>::sort(  
        struct Files *fList,
        struct params *p,
 unsigned int blockscount, unsigned int threads, unsigned int sbsize, unsigned int phase)
{
 unsigned int size=p->nG;
 unsigned int nE = p->nE;
 unsigned int * data = new unsigned int[size];
        float * dataf = new float[size];
 unsigned int * pos = new unsigned int[size]; 
 float * cum = new float[size];
 double medida, timerValue[5];	
 
 int exp,i;
 
	if(!init)
		return 1;

	if(!threads||!blockscount||!sbsize)
	{
		threads   = 1<<(int)round(log(size * TK + TM)/log(2.0));
		blockscount = 1<<(int)round(log(size * MK + MM)/log(2.0));
		sbsize    = 1<<(int)round(log(size * SK + SM)/log(2.0));
	}
 

#ifdef HASATOMICS
		unsigned int* doh;
		unsigned int oh;

		hipGetSymbolAddress((void**)&doh,"ohtotal");
		oh=0;
		hipMemcpy(doh,&oh,4,hipMemcpyHostToDevice);
#endif

	if(threads>MAXTHREADS)
		return 1; 

	if(blockscount>MAXBLOCKS)
		return 1;

timerValue[0]=0.0;
timerValue[1]=0.0;
timerValue[2]=0.0;
timerValue[3]=0.0;
timerValue[4]=0.0;
  

SimpleTimer st,st1,st2;
int device;
if(p->GPU==-1)
{
//MRequena: Displaying available devices visible to host
int deviceCount;
hipGetDeviceCount (&deviceCount);
printf("Available devices:\n");
for (device=0; device<deviceCount; device++){
hipDeviceProp_t deviceProp;
hipGetDeviceProperties(&deviceProp, device);
printf("\tDevice %d: %s \n",device,deviceProp.name);
}
device= -1;
while (device<0 || device>=deviceCount){
printf("\nIntrduce un dispositivo: ");
scanf("%d", &device);
}
}
else  device = p->GPU;
printf("\nUsing GPU device %d.\n",device);
   hipSetDevice(device);    // CUT_DEVICE_INIT();


	// Copy the data to the graphics card and create an auxiallary array
	ddata2 = 0; ddata = 0; dcum =0;
        dpos =0; dpos2=0;  //andres
	if(!errCheck(hipMalloc((void**)&ddata2,(size)*sizeof(element))))
		return 1;
	if(!errCheck(hipMalloc((void**)&ddata,(size)*sizeof(element))))
		return 1;

        // andres prepare indices
	if(!errCheck(hipMalloc((void**)&dpos,(size)*sizeof(unsigned int))))
		return 1;
	if(!errCheck(hipMalloc((void**)&dpos2,(size)*sizeof(unsigned int))))
		return 1;
	
 	// init mean vector to zero values
        if(!errCheck(hipMalloc((void**)&dcum,(size)*sizeof(float))))
		return 1;

	if(!errCheck(hipMemset(dcum, 0, size*sizeof(float)) ))
		return 1;
		
	// abre fichero
	FILE * fI;
	char nombref[1024];
		
	if((fI=fopen(INDICES_TMP,"wb"))==NULL) return 1;
		
	

// bucle de experimentos
for(exp=0; exp<nE; exp++)
{

// carga fichero
  	
  	// Start measuring time
		hipDeviceSynchronize();
		
		st.start();
    printf("Loading %3d/%d File: %s Genes: %d\n",exp,nE,fList[exp].fname,size);
	
    LoadFile(fList, exp, data);
    
    // Measure the time taken by loading from HD
		timerValue[1]+= st.end();
		
		st.start();
	
 	
	if(!errCheck(hipMemcpy(ddata, data, size*sizeof(element), hipMemcpyHostToDevice) ))
		return 1;

    // init pos vector
        unsigned int k;
        for(k=0; k<size; k++) {pos[k]=k;}  // hacer esto en paralelo en la GPU ?
    	if(!errCheck(hipMemcpy(dpos, pos, size*sizeof(unsigned int), hipMemcpyHostToDevice) )) return 1;

        // init_pos <<<blockscount, threads>>> (dpos, size);   // version GPU
        
        // Measure the time taken by loading into device
		timerValue[2]+= st.end();
	

	
	
			// Start measuring time
		
		st.start();
	

	// We start with a set containg only the sequence to be sorted
	// This will grow as we partition the data
	workset[0].beg = 0;
	workset[0].end = size;
	workset[0].orgbeg = 0;
	workset[0].orgend = size;
	workset[0].altered = false;
	workset[0].flip = false;


	// Get a starting pivot
	workset[0].pivot = (min(min(data[0],data[size/2]),data[size-1]) + max(max(data[0],data[size/2]),data[size-1]))/2;
	unsigned int worksize = 1;


	unsigned int blocks = blockscount/2;
	unsigned totsize = size;
	unsigned int maxlength = (size/blocks)/4;

	unsigned int iterations = 0;
	bool flip = true;


	// Partition the sequences until we have enough
	while(worksize<blocks)
	{

		unsigned int ws = totsize/blocks;
		unsigned int paramsize = 0;
		// Go through the sequences we have and divide them into sections
		// and assign thread blocks according to their size

		for(unsigned int i=0;i<worksize;i++)
		{

			if((workset[i].end-workset[i].beg)<maxlength)
				continue;

			// Larger sequences gets more thread blocks assigned to them
            unsigned int blocksassigned = max((workset[i].end-workset[i].beg)/ws,1);

			for(unsigned int q=0;q<blocksassigned;q++)
			{
				params[paramsize].from = workset[i].beg + ws*q;
				params[paramsize].end = params[paramsize].from + ws;
				params[paramsize].pivot = workset[i].pivot;
				params[paramsize].ptr = i;
				params[paramsize].last = false;
				paramsize++;
				
			}
			params[paramsize-1].last = true;
			params[paramsize-1].end = workset[i].end;

			workset[i].lmaxpiv=0;
			workset[i].lminpiv=0xffffffff;
			workset[i].rmaxpiv=0;
			workset[i].rminpiv=0xffffffff;
		}

		if(paramsize==0)
			break;
        		// Copy the block assignment to the GPU
		if(!errCheck(hipMemcpy(dparams, params, paramsize*sizeof(Params<element>), hipMemcpyHostToDevice) ))
			return 1;
 if(p->Verbose) printf("part1\n");

		// Do the cumulative sum
		if(flip)
			part1<<< paramsize, THREADS, (THREADS+1)*2*4+THREADS*2*4 >>>(ddata,dparams,dhists,dlength);
		else
			part1<<< paramsize, THREADS, (THREADS+1)*2*4+THREADS*2*4 >>>(ddata2,dparams,dhists,dlength);
		if(!errCheck((hipMemcpy(length, dlength,sizeof(Length<element>) , hipMemcpyDeviceToHost) )))
			return 1; 

		// Do the block cumulative sum. Done on the CPU since not all cards have support for
		// atomic operations yet. 
		for(unsigned int i=0;i<paramsize;i++)
		{
			unsigned int l = length->left[i];
			unsigned int r = length->right[i];
			
			length->left[i] = workset[params[i].ptr].beg;
			length->right[i] = workset[params[i].ptr].end;
			
			workset[params[i].ptr].beg+=l;
			workset[params[i].ptr].end-=r;
			workset[params[i].ptr].altered = true;
			
			workset[params[i].ptr].rmaxpiv = max(length->maxpiv[i],workset[params[i].ptr].rmaxpiv);
			workset[params[i].ptr].lminpiv = min(length->minpiv[i],workset[params[i].ptr].lminpiv);
			
			workset[params[i].ptr].lmaxpiv = min(workset[params[i].ptr].pivot,workset[params[i].ptr].rmaxpiv); 
			workset[params[i].ptr].rminpiv = max(workset[params[i].ptr].pivot,workset[params[i].ptr].lminpiv); 

			
		}

		// Copy the result of the block cumulative sum to the GPU
		if(!errCheck((hipMemcpy(dlength, length, sizeof(Length<element>), hipMemcpyHostToDevice) )))
			return 1;

		// Move the elements to their correct position
 if(p->Verbose) printf("part2\n");
		if(flip)
			part2<<< paramsize, THREADS >>>(dpos,dpos2,ddata,ddata2,dparams,dhists,dlength);
		else
			part2<<< paramsize, THREADS >>>(dpos2,dpos,ddata2,ddata,dparams,dhists,dlength);

		// Fill in the pivot value between the left and right blocks
		//part3<<< paramsize, THREADS >>>(ddata,dparams,dhists,dlength);

		flip = !flip;

		// Add the sequences resulting from the partitioning
		// to set
		unsigned int oldworksize = worksize;
		totsize = 0;
		for(unsigned int i=0;i<oldworksize;i++)
		{
			if(workset[i].altered)
			{
				if(workset[i].beg-workset[i].orgbeg>=maxlength)
					totsize += workset[i].beg-workset[i].orgbeg;
				if(workset[i].orgend-workset[i].end>=maxlength)
					totsize += workset[i].orgend-workset[i].end;

				workset[worksize].beg = workset[worksize].orgbeg = workset[i].orgbeg;
				workset[worksize].end = workset[worksize].orgend = workset[i].beg;
				workset[worksize].flip=flip;
				workset[worksize].altered = false;
				workset[worksize].pivot = (workset[i].lminpiv/2+workset[i].lmaxpiv/2);

				worksize++;

				workset[i].orgbeg = workset[i].beg = workset[i].end;
				workset[i].end = workset[i].orgend;
				workset[i].flip=flip;
				workset[i].pivot = (workset[i].rminpiv/2+workset[i].rmaxpiv/2);
				workset[i].altered = false;
			}
		}
		iterations++;

	}

	// Due to the poor scheduler on some graphics card
	// we need to sort the order in which the blocks
	// are sorted to avoid poor scheduling decisions
	unsigned int sortblocks[MAXBLOCKS*2];
	for(int i=0;i<worksize;i++)
		sortblocks[i]=((workset[i].end-workset[i].beg)<<(int)round(log((float)(MAXBLOCKS*4.0f))/log(2.0f))) + i;
	std::sort(&sortblocks[0],&sortblocks[worksize]);

	if(worksize!=0)
	{
		// Copy the block assignments to the GPU
		for(int i=0;i<worksize;i++)
		{
		 	unsigned int q = (worksize-1)-(sortblocks[i]&(MAXBLOCKS*4-1));

			lqparams[i].beg =  workset[q].beg;
			lqparams[i].end = workset[q].end;
			lqparams[i].flip = workset[q].flip;
			lqparams[i].sbsize = sbsize;
//                      printf("BEFS: %3d %10d %10d %10d %10d\n",i,lqparams[i].beg,lqparams[i].end,lqparams[i].flip,sbsize);
		}

		if(!errCheck((hipMemcpy(dlqparams, lqparams, worksize*sizeof(LQSortParams), hipMemcpyHostToDevice) )))
			return 1;
                
 if(p->Verbose) printf("lqsort\n");
		// Run the local quicksort, the one that doesn't need inter-block synchronization
		if(phase!=1)
			lqsort<<< worksize, THREADS >>>(dpos,dpos2,ddata,ddata2,dlqparams,phase);
	
	}

	hipDeviceSynchronize();
	
 if(p->Verbose) printf("promedio .......... %d\n",exp);
        promedio <<<blockscount, threads>>> (ddata, dcum, size);
        
    // Measure the Time taken by CPU+GPU
                medida= st.end();
		timerValue[0]+=medida; 
                printf("processing time: %8.2f ms\n",medida);
	
	st.start();    
    // lee posiciones y escribelas a disco    
    if(!errCheck((hipMemcpy(pos, dpos, size*sizeof(unsigned int), hipMemcpyDeviceToHost) )))		return 1;
    
    // Measure the time taken by loading back into CPU
		timerValue[4]+= st.end();
	
	// write to pos file
	    st.start();    
	     
          //fseek(fI, size*exp*sizeof(unsigned int), SEEK_SET);
        fwrite(pos, sizeof(unsigned int), size, fI);    
	    
    // Measure the time taken by loading back into CPU
		timerValue[3]+= st.end();
	            
        
     } // end bucle exp   
     
     fclose(fI);
       
       
 if(p->Verbose) printf("final divide .......... \n");
        divide   <<<blockscount, threads>>> (dcum, nE, size);
 
	
	

	err = hipDeviceSynchronize();
	// Free the data
	if(err!=hipSuccess)
	{
		hipFree(ddata);
		hipFree(ddata2);
		hipFree(dpos2);
		hipFree(dpos);
		hipFree(dcum);
		return 1;
	}
	// Copy the result back to the CPU
	//if(!errCheck((hipMemcpy(data, ddata, size*sizeof(element), hipMemcpyDeviceToHost) )))		return 1;
	
     st.start();	
	if(!errCheck((hipMemcpy(cum, dcum, size*sizeof(element), hipMemcpyDeviceToHost) )))
		return 1;
		
		// Measure the time taken by loading back into CPU
		timerValue[4]+= st.end();
		
    
    
    FILE * fO;
    
    // write to OUTPUT file
	st.start();    
	
	if((fI=fopen(INDICES_TMP,"rb"))==NULL) return 1;
	if((fO=fopen(p->fOutName,"wb"))==NULL) return 1;

	for(exp=0;exp<nE;exp++)
	{
	  fread(pos, sizeof(unsigned int), size, fI);    
	  for(i=0;i<size;i++)
	  {
	   dataf[i]=cum[pos[i]];
	  }	
	  fwrite(dataf,sizeof(float), size, fO);
if(p->Verbose)
{
		for(int i=0;i<10;i++)
		printf("data  ::: %5d = %10.2f   ,   %5d = %10.2f   ,    %5d = %10.2f\n",i,dataf[i],size/2-5+i,dataf[size/2-5+i],size-10+i,dataf[size-10+i]);
	 printf("----------\n\n");	
}
if(p->Verbose)
{
		for(int i=0;i<10;i++)
		printf("pos  ::: %5d = %10d   ,   %5d = %10d   ,    %5d = %10d\n",i,pos[i],size/2-5+i,pos[size/2-5+i],size-10+i,pos[size-10+i]);
	 printf("----------\n\n");	
}
	  
	}
	
	fclose(fI); fclose(fO);
	    
	    
    // Measure the time taken by loading back into CPU
		timerValue[3]+= st.end();
	
	
	
	//debug salida
if(0 && p->Verbose)
{
		for(int i=0;i<1000000;i++)
		//printf("cum  ::: %5d = %10.2f   ,   %5d = %10.2f   ,    %5d = %10.2f\n",i,cum[i],size/2-500+i,cum[size/2-500+i],size-1000+i,cum[size-1000+i]);
		printf("%10.2f,%10.2f,%10.2f,%10.2f,%10.2f,%10.2f\n",cum[i],cum[i]+1000000,cum[i]+2000000, cum[i]+3000000, cum[i]+4000000, cum[i]+5000000 );
		
}
		printf("\n----------  TIEMPOS -----------------------\n");
	printf("CPU+GPU  : %10.2f\n",timerValue[0]);
	printf("HD->CPU  : %10.2f\n",timerValue[1]);
	printf("CPU->DEV : %10.2f\n",timerValue[2]);
	printf("DEV->CPU : %10.2f\n",timerValue[4]);
	printf("CPU->HD  : %10.2f\n",timerValue[3]);
	    printf("--------------------------------------------\n");
    printf("TOTAL    : %10.2f\n\n",timerValue[0]+timerValue[1]+timerValue[2]+timerValue[4]+timerValue[3]);
	
	
	hipFree(ddata);
	hipFree(ddata2);
		hipFree(dpos2);
		hipFree(dpos);
		hipFree(dcum);
 
	return 0;
}

template <typename element>
bool GPUQSort<element>::errCheck(int e)
{
	if(e==hipSuccess)
		return true;

	err = e;
	hipFree(ddata);
	hipFree(ddata2);
	return false;
}

template <typename element>
GPUQSort<element>::GPUQSort():init(false),workset(0),params(0),length(0),lqparams(0),dlqparams(0),
							  dhists(0),dlength(0),dparams(0)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	if(!strcmp(deviceProp.name,"GeForce 8800 GTX"))
	{
		TK = 1.17125033316e-005f;
		TM = 52.855721393f;
		MK = 3.7480010661e-005f;
		MM = 476.338308458f;
		SK = 4.68500133262e-005f;
		SM = 211.422885572f;
	}
	else
	if(!strcmp(deviceProp.name,"GeForce 8600 GTS"))
	{
		TK = 0.0f;
		TM = 64.0f;
		MK = 0.0000951623403898f;
		MM = 476.338308458f;
		SK = 0.0000321583081317f;
		SM = 202.666666667f;
	}
	else
	{
		TK = 0;
		TM = 128;
		MK = 0;
		MM = 512;
		SK = 0;
		SM = 512;
	}

	if(hipHostMalloc((void**)&workset,MAXBLOCKS*2*sizeof(BlockSize<element>))!=hipSuccess) return;
	if(hipHostMalloc((void**)&params,MAXBLOCKS*sizeof(Params<element>))!=hipSuccess) return;
	if(hipHostMalloc((void**)&length,sizeof(Length<element>))!=hipSuccess) return;
	if(hipHostMalloc((void**)&lqparams,MAXBLOCKS*sizeof(LQSortParams))!=hipSuccess) return;
	if(hipMalloc((void**)&dlqparams,MAXBLOCKS*sizeof(LQSortParams))!=hipSuccess) return;
	if(hipMalloc((void**)&dhists,sizeof(Hist))!=hipSuccess) return;
	if(hipMalloc((void**)&dlength,sizeof(Length<element>))!=hipSuccess) return;
	if(hipMalloc((void**)&dparams,MAXBLOCKS*sizeof(Params<element>))!=hipSuccess) return;

	init = true;
}

/**
* Returns the latest error message
* @returns the latest error message
*/
template <typename element>
const char* GPUQSort<element>::getErrorStr()
{
	return hipGetErrorString((hipError_t)err);
}

template <typename element>
GPUQSort<element>::~GPUQSort()
{
	hipHostFree(workset);
	hipHostFree(params);
	hipHostFree(length);
	hipHostFree(lqparams);
	hipFree(dparams);
	hipFree(dlqparams);
	hipFree(dhists);
	hipFree(dlength);
}

// Exported functions

char* expErrMsg = "No errors";

 GPUQSort<unsigned int>* s=0;

extern "C" 
int gpuqsort( struct Files *fList,
        struct params *p, unsigned int blockscount, unsigned int threads, unsigned int sbsize, unsigned int phase)
{
	if(s==0)
		s=new GPUQSort<unsigned int>();


	if(s->sort(fList,p, blockscount, threads, sbsize, phase)!=0)
	{
		expErrMsg = (char*)s->getErrorStr();
		return 1;
	}
	else
		return 0;
}


extern "C"
const char* getGPUSortErrorStr()
{
	return expErrMsg;
}
